#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include "summation.cu"

#define BLOCK_SIZE 256 // Number of threads per block

__global__ void mean(int *input, int *output, int len) {
    __shared__ int shared[BLOCK_SIZE];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    shared[tid] = (i < len) ? input[i] : 0;
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared[tid] += shared[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) {
        output[blockIdx.x] = shared[0] / len;
    }
}

// void mean(int *input, int *output, int len) {
//     int *d_input, *d_sum, sumResult;
//     int blockSize = BLOCK_SIZE;
//     int numBlocks = (len + blockSize - 1) / blockSize;

//     hipMalloc(&d_input, len*sizeof(int));
//     hipMalloc(&d_sum, sizeof(int));

//     hipMemcpy(d_input, input, len*sizeof(int), hipMemcpyHostToDevice);

//     summation<<<numBlocks, BLOCK_SIZE>>>(d_input, d_sum, len);
//     hipDeviceSynchronize();

//     hipMemcpy(&sumResult, d_sum, sizeof(int), hipMemcpyDeviceToHost);

//     *output = static_cast<float>(sumResult) / len;

//     hipFree(d_input);
//     hipFree(d_sum);
// }

// __global__ void mean(int *input, int *output, int len) {
//     calculateMean<<<1, BLOCK_SIZE>>>(input, output, len);
// }


int main(int argc, char *argv[]) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <element1> <element2> ... <elementN>" << std::endl;
        return 1;
    }

    int N = argc - 1;
    int *input = new int[N];
    for (int i = 0; i < N; ++i) {
        input[i] = std::strtol(argv[i + 1], nullptr, 10);
    }

    int *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(int));
    hipMalloc(&d_output, sizeof(int));

    hipMemcpy(d_input, input, N * sizeof(int), hipMemcpyHostToDevice);

    // Assuming N is not too large for a single block
    mean<<<1, BLOCK_SIZE>>>(d_input, d_output, N);

    int result;
    hipMemcpy(&result, d_output, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Mean: " << result << std::endl;

    delete[] input;

    return 0;
}
