#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>
#include <iomanip>
#include <cstdlib>
#include "dotprod.cuh"

#define TILE_WIDTH 16

// Function to generate a matrix with random values
std::vector<std::vector<float>> generateRandomMatrix(int rows, int cols) {
    std::vector<std::vector<float>> matrix(rows, std::vector<float>(cols));
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(-1.0, 1.0);

    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            matrix[i][j] = dist(gen);
        }
    }

    return matrix;
}

// Function to print a matrix
void printMatrix(const std::vector<std::vector<float>>& matrix) {
    for (const auto &row : matrix) {
        for (auto elem : row) {
            std::cout << std::fixed << std::setprecision(4) << elem << " ";
        }
        std::cout << std::endl;
    }
}

float* flattenMatrix(const std::vector<std::vector<float>>& matrix, int rows, int cols) {
    float* flat = new float[rows * cols];
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            flat[i * cols + j] = matrix[i][j];
        }
    }
    return flat;
}


// __global__ void matmul(float* A, float* B, float* C, int ARows, int ACols, int BCols) {
//     int row = blockIdx.y * blockDim.y + threadIdx.y;
//     int col = blockIdx.x * blockDim.x + threadIdx.x;

//     if(row < ARows && col < BCols) {
//         float sum = 0.0;
//         for (int k = 0; k < ACols; ++k) {
//             sum += A[row * ACols + k] * B[k * BCols + col];
//         }
//         C[row * BCols + col] = sum;
//     }
// }

__global__ void matmul(float* A, float* B, float* C, int ARows, int ACols, int BCols) {
    __shared__ float Asub[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bsub[TILE_WIDTH][TILE_WIDTH];
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    float sum = 0.0;
    // Loop over sub-matrices of A and B that are required to compute the C element
    for (int k = 0; k < (ACols + TILE_WIDTH - 1) / TILE_WIDTH; k++) {
        // Load sub-matrices into shared memory
        if (k*TILE_WIDTH + threadIdx.x < ACols && row < ARows)
            Asub[threadIdx.y][threadIdx.x] = A[row*ACols + k*TILE_WIDTH + threadIdx.x];
        else
            Asub[threadIdx.y][threadIdx.x] = 0.0;

        if (k*TILE_WIDTH + threadIdx.y < BCols && col < BCols)
            Bsub[threadIdx.y][threadIdx.x] = B[(k*TILE_WIDTH + threadIdx.y)*BCols + col];
        else
            Bsub[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();

        for (int n = 0; n < TILE_WIDTH; ++n)
            sum += Asub[threadIdx.y][n] * Bsub[n][threadIdx.x];

        __syncthreads();
    }
    
    if (row < ARows && col < BCols)
        C[row*BCols + col] = sum;
}


#ifdef COMPILE_MAIN
int main(int argc, char* argv[]) {
    if (argc != 4) {
        std::cerr << "Usage: " << argv[0] << " <m> <n> <p>\n";
        return 1;
    }

    int m = std::atoi(argv[1]);
    int n = std::atoi(argv[2]);
    int p = std::atoi(argv[3]);

    auto matrixA = generateRandomMatrix(m, n);
    auto matrixB = generateRandomMatrix(n, p);

    std::cout << "Matrix A (" << m << "x" << n << "):\n";
    printMatrix(matrixA);
    std::cout << "\nMatrix B (" << n << "x" << p << "):\n";
    printMatrix(matrixB);

        // Flatten matrices
    float* A_flat = flattenMatrix(matrixA, m, n);
    float* B_flat = flattenMatrix(matrixB, n, p);
    float* C_flat = new float[m * p];

    // CUDA memory allocation
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, m * n * sizeof(float));
    hipMalloc(&d_B, n * p * sizeof(float));
    hipMalloc(&d_C, m * p * sizeof(float));

    hipMemcpy(d_A, A_flat, m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B_flat, n * p * sizeof(float), hipMemcpyHostToDevice);

    //const int TILE_WIDTH = 16;
    dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 blocksPerGrid((p + TILE_WIDTH - 1) / TILE_WIDTH, (m + TILE_WIDTH - 1) / TILE_WIDTH);

    matmulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, m, n, p);

    hipMemcpy(C_flat, d_C, m * p * sizeof(float), hipMemcpyDeviceToHost);

    std::vector<std::vector<float>> matrixC(m, std::vector<float>(p));
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < p; ++j) {
            matrixC[i][j] = C_flat[i * p + j];
        }
    }
    std::cout << "\nResult Matrix C (" << m << "x" << p << "):\n";
    printMatrix(matrixC);

    delete[] A_flat;
    delete[] B_flat;
    delete[] C_flat;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
#endif
