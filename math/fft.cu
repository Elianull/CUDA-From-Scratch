#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

const float PI = 2*acos(0.0);

__global__ void fftBreakdown(hipFloatComplex* input, hipFloatComplex* output, int size) {
    extern __shared__ hipFloatComplex sharedInput[];
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int numPairs = (size + 1) / 2;
    int idx = bid * blockDim.x + tid;

    if (idx < numPairs) {
        sharedInput[tid] = input[idx];
        __syncthreads();

        output[idx] = sharedInput[tid];
        if (idx + numPairs < size) {
            output[idx + numPairs] = input[idx + numPairs];
        }
    }
}

std::vector<std::vector<hipFloatComplex>> fftBreakdownHost(std::vector<hipFloatComplex>& input) {
    int size = input.size();
    int numPairs = (size + 1) / 2;
    bool oddSize = (size % 2 != 0);
    std::vector<std::vector<hipFloatComplex>> output(numPairs, std::vector<hipFloatComplex>(2));

    // Pad the input with a zero element if the size is odd
    if (oddSize) {
        input.push_back(make_hipFloatComplex(0.0f, 0.0f));
        size++;
    }

    hipFloatComplex* d_input;
    hipFloatComplex* d_output;
    hipMalloc(&d_input, size * sizeof(hipFloatComplex));
    hipMalloc(&d_output, size * sizeof(hipFloatComplex));

    hipMemcpy(d_input, input.data(), size * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (numPairs + blockSize - 1) / blockSize;
    fftBreakdown<<<numBlocks, blockSize, blockSize * sizeof(hipFloatComplex)>>>(d_input, d_output, size);
    hipDeviceSynchronize();

    for (int i = 0; i < numPairs; i++) {
        hipMemcpy(&output[i][0], &d_output[i], sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
        hipMemcpy(&output[i][1], &d_output[i + numPairs], sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    }

    hipFree(d_input);
    hipFree(d_output);

    return output;
}

// __global__ void dftKernel(hipFloatComplex* input, hipFloatComplex* output, int size) {
//     int tid = threadIdx.x;
//     int bid = blockIdx.x;
//     int idx = bid * blockDim.x + tid;
//     int numPairs = size / 2;

//     if (idx < numPairs) {
//         hipFloatComplex x = input[idx*2];
//         hipFloatComplex y = input[idx*2 + 1];

//         float angle = -2.0f * M_PI * idx / numPairs;
//         hipFloatComplex twiddle = make_hipFloatComplex(cos(angle), sin(angle));

//         output[idx*2] = hipCaddf(x, hipCmulf(y, twiddle));
//         output[idx*2 + 1] = hipCsubf(x, hipCmulf(y, twiddle));
//     }
// }

// __global__ void dftKernel(hipFloatComplex* d_input, hipFloatComplex* d_output, int N) {
//     int idx = threadIdx.x + blockIdx.x * blockDim.x;

//     if (idx < N/2) {
//         hipFloatComplex x = d_input[idx * 2];
//         hipFloatComplex y = d_input[idx * 2 + 1];

//         hipFloatComplex sum_x = make_hipFloatComplex(0.0f, 0.0f);
//         hipFloatComplex sum_y = make_hipFloatComplex(0.0f, 0.0f);

//         for (int n = 0; n < 2; n++) {
//             float angle_x = -2.0f * M_PI * 0 * n / 2.0f;
//             float angle_y = -2.0f * M_PI * 1 * n / 2.0f;

//             hipFloatComplex exp_term_x = make_hipFloatComplex(cosf(angle_x), sinf(angle_x));
//             hipFloatComplex exp_term_y = make_hipFloatComplex(cosf(angle_y), sinf(angle_y));

//             hipFloatComplex product_x = hipCmulf(x, exp_term_x);
//             hipFloatComplex product_y = hipCmulf(y, exp_term_y);

//             sum_x = hipCaddf(sum_x, product_x);
//             sum_y = hipCaddf(sum_y, product_y);
//         }

//         d_output[idx * 2] = sum_x;
//         d_output[idx * 2 + 1] = sum_y;
//     }
// }

__global__ void dft_kernel(hipFloatComplex* input, hipFloatComplex* output, uint32_t N, uint32_t numPairs)
{
    // Find which element of Y this thread is computing
    int k = threadIdx.x + blockIdx.x * blockDim.x;
    int p = threadIdx.y + blockIdx.y * blockDim.y;
    
    if (k < N && p < numPairs) {
        hipFloatComplex sum = make_hipFloatComplex(0, 0);
        
        // Save the value of -2 pi * k / N
        float c = -2 * M_PI * k / N;
        
        // Each thread computes a summation containing N terms
        for (size_t n = 0; n < N; n++) {
            // e^ix = cos x + i sin x
            // Compute x[n] * exp(-2i pi * k * n / N)
            float ti, tr;
            sincosf(c * n, &ti, &tr);
            sum = hipCaddf(sum, hipCmulf(input[p * N + n], make_hipFloatComplex(tr, ti)));
        }
        
        output[p * N + k] = sum;
    }
}

// This function computes the DFT on the GPU.
void performDFT(hipFloatComplex* input, hipFloatComplex* output, uint32_t N, uint32_t numPairs)
{
    hipFloatComplex* d_input;
    hipFloatComplex* d_output;
    
    hipMalloc((void**)&d_output, sizeof(hipFloatComplex) * N * numPairs);
    hipMalloc((void**)&d_input, sizeof(hipFloatComplex) * N * numPairs);
    
    hipMemcpy(d_input, input, sizeof(hipFloatComplex) * N * numPairs, hipMemcpyHostToDevice);
    
    int cuda_device_ix = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, cuda_device_ix);
    
    // One thread for each element of the output vector
    int block_size_x = min(N, prop.maxThreadsDim[0]);
    int block_size_y = min(numPairs, prop.maxThreadsDim[1]);
    int grid_size_x = (N + block_size_x - 1) / block_size_x;
    int grid_size_y = (numPairs + block_size_y - 1) / block_size_y;
    
    dim3 block(block_size_x, block_size_y);
    dim3 grid(grid_size_x, grid_size_y);
    
    dft_kernel<<<grid, block>>>(d_input, d_output, N, numPairs);
    
    hipMemcpy(output, d_output, sizeof(hipFloatComplex) * N * numPairs, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

std::vector<hipFloatComplex> computeFFT(std::vector<hipFloatComplex>& input) {
    int size = input.size();
    std::vector<hipFloatComplex> output(size);

    hipFloatComplex* d_input;
    hipMalloc(&d_input, size*sizeof(hipFloatComplex));

    hipMemcpy(d_input, input.data(), size * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    std::cout << "Input: ";
    for (int i = 0; i < size; i++) {
        std::cout << "(" << hipCrealf(input[i]) << ", " << hipCimagf(input[i]) << "i) ";
    }
    std::cout << std::endl;

    std::vector<std::vector<hipFloatComplex>> pairs = fftBreakdownHost(input);
    std::cout << "FFT Breakdown:" << std::endl;
    for (int i = 0; i < pairs.size(); i++) {
        std::cout << "Pair " << i << ": (" << hipCrealf(pairs[i][0]) << ", " << hipCimagf(pairs[i][0]) << "i) and ("
                  << hipCrealf(pairs[i][1]) << ", " << hipCimagf(pairs[i][1]) << "i)" << std::endl;
    }
    
        int N = 2;  // Assuming each pair has 2 elements
    int numPairs = pairs.size();

    hipFloatComplex* dftOutput = new hipFloatComplex[N * numPairs];

    // Convert the pairs vector to a flattened array
    hipFloatComplex* flatPairs = new hipFloatComplex[N * numPairs];
    for (int i = 0; i < numPairs; i++) {
        flatPairs[i * N + 0] = pairs[i][0];
        flatPairs[i * N + 1] = pairs[i][1];
    }

    performDFT(flatPairs, dftOutput, N, numPairs);

    std::cout << "DFT Outputs: ";
    for (int i = 0; i < N * numPairs; i++) {
        std::cout << "(" << hipCrealf(dftOutput[i]) << ", " << hipCimagf(dftOutput[i]) << "i) ";
    }
    std::cout << std::endl;

    return input;
}

#ifdef COMPILE_MAIN
int main(int argc, char *argv[]) {
    if (argc <= 1 || argc % 2 != 1) {
        std::cout << "Usage: " << argv[0] << " <real1> <imaginary1> <real2> <imaginary2> ... <realN> <imaginaryN>\n";
        return 1;
    }

    int size = (argc - 1) / 2;
    size_t bytes = size * sizeof(float);

    float *real_values = (float*)malloc(bytes);
    float *imaginary_values = (float*)malloc(bytes);
    std::vector<hipFloatComplex> complex_values(size);

    for (int i = 0; i < 2 * size; i += 2) {
        int index = i / 2;
        complex_values[index] = make_hipFloatComplex(std::atof(argv[1 + i]), std::atof(argv[2 + i]));
    }
    
    std::vector<hipFloatComplex> fft_result = computeFFT(complex_values);

    return 0;
}
#endif
