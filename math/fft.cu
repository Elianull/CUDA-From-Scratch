#include <iostream>
#include <cstdlib>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

const float PI = 2*acos(0.0);

__global__ void fftBreakdown(hipFloatComplex* input, hipFloatComplex* output, int size) {
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int numPairs = (size+1) / 2;
    
    if (bid * blockDim.x + tid < numPairs) {
        int idx = bid * blockDim.x + tid;
        output[idx] = input[idx];
        output[idx + numPairs] = input[idx + numPairs];
    }
}

std::vector<std::vector<hipFloatComplex>> fftBreakdownHost(std::vector<hipFloatComplex>& input) {
    int size = input.size();
    int numPairs = (size + 1) / 2;
    bool oddSize = (size % 2 != 0);
    std::vector<std::vector<hipFloatComplex>> output(numPairs, std::vector<hipFloatComplex>(2));

    // Pad the input with a zero element if the size is odd
    if (oddSize) {
        input.push_back(make_hipFloatComplex(0.0f, 0.0f));
        size++;
    }

    // Allocate device memory
    hipFloatComplex* d_input;
    hipFloatComplex* d_output;
    hipMalloc(&d_input, size * sizeof(hipFloatComplex));
    hipMalloc(&d_output, size * sizeof(hipFloatComplex));

    // Copy input data to device
    hipMemcpy(d_input, input.data(), size * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    // Launch the fftBreakdown kernel
    int blockSize = 256;
    int numBlocks = (numPairs + blockSize - 1) / blockSize;
    fftBreakdown<<<numBlocks, blockSize>>>(d_input, d_output, size);
    hipDeviceSynchronize();  // Wait for kernel to finish

    // Copy the pairs back to the host
    for (int i = 0; i < numPairs; i++) {
        hipMemcpy(&output[i][0], &d_output[i], sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
        hipMemcpy(&output[i][1], &d_output[i + numPairs], sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    }

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);

    return output;
}

std::vector<hipFloatComplex> computeFFT(std::vector<hipFloatComplex>& input) {
    int size = input.size();
    std::vector<hipFloatComplex> output(size);

    hipFloatComplex* d_input;
    hipMalloc(&d_input, size*sizeof(hipFloatComplex));

    hipMemcpy(d_input, input.data(), size * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    for (int i = 0; i < size; i++) {
        std::cout << "(" << hipCrealf(input[i]) << ", " << hipCimagf(input[i]) << "i) ";
    }
    std::cout << std::endl;

    std::vector<std::vector<hipFloatComplex>> pairs = fftBreakdownHost(input);
    std::cout << "FFT Breakdown:" << std::endl;
    for (int i = 0; i < pairs.size(); i++) {
        std::cout << "Pair " << i << ": (" << hipCrealf(pairs[i][0]) << ", " << hipCimagf(pairs[i][0]) << "i) and ("
                  << hipCrealf(pairs[i][1]) << ", " << hipCimagf(pairs[i][1]) << "i)" << std::endl;
    }
    
    return input;
}

#ifdef COMPILE_MAIN
int main(int argc, char *argv[]) {
    if (argc <= 1 || argc % 2 != 1) {
        std::cout << "Usage: " << argv[0] << " <real1> <imaginary1> <real2> <imaginary2> ... <realN> <imaginaryN>\n";
        return 1;
    }

    int size = (argc - 1) / 2;
    size_t bytes = size * sizeof(float);

    float *real_values = (float*)malloc(bytes);
    float *imaginary_values = (float*)malloc(bytes);
    std::vector<cuFloatComplex> complex_values(size);

    for (int i = 0; i < 2 * size; i += 2) {
        int index = i / 2;
        complex_values[index] = make_cuFloatComplex(std::atof(argv[1 + i]), std::atof(argv[2 + i]));
    }
    
    std::vector<cuFloatComplex> fft_result = computeFFT(complex_values);

    return 0;
}
#endif
