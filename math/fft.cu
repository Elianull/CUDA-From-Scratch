#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#ifndef M_PI
#define M_PI (3.14159265358979323846)
#endif

__global__ void fftBreakdown(hipFloatComplex* input, hipFloatComplex* output, int size) {
    extern __shared__ hipFloatComplex sharedInput[];
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int numPairs = (size + 1) / 2;
    int idx = bid * blockDim.x + tid;

    if (idx < numPairs) {
        sharedInput[tid] = input[idx];
        __syncthreads();

        output[idx] = sharedInput[tid];
        if (idx + numPairs < size) {
            output[idx + numPairs] = input[idx + numPairs];
        }
    }
}

std::vector<std::vector<hipFloatComplex>> fftBreakdownHost(std::vector<hipFloatComplex>& input) {
    int size = input.size();
    int numPairs = (size + 1) / 2;
    bool oddSize = (size % 2 != 0);
    std::vector<std::vector<hipFloatComplex>> output(numPairs, std::vector<hipFloatComplex>(2));

    // Pad the input with a zero element if the size is odd
    if (oddSize) {
        input.push_back(make_hipFloatComplex(0.0f, 0.0f));
        size++;
    }

    hipFloatComplex* d_input;
    hipFloatComplex* d_output;
    hipMalloc(&d_input, size * sizeof(hipFloatComplex));
    hipMalloc(&d_output, size * sizeof(hipFloatComplex));

    hipMemcpy(d_input, input.data(), size * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (numPairs + blockSize - 1) / blockSize;
    fftBreakdown<<<numBlocks, blockSize, blockSize * sizeof(hipFloatComplex)>>>(d_input, d_output, size);
    hipDeviceSynchronize();

    for (int i = 0; i < numPairs; i++) {
        hipMemcpy(&output[i][0], &d_output[i], sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
        hipMemcpy(&output[i][1], &d_output[i + numPairs], sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    }

    hipFree(d_input);
    hipFree(d_output);

    return output;
}

__global__ void dft_kernel(hipFloatComplex* input, hipFloatComplex* output, uint32_t N, uint32_t numPairs)
{
    // Find which element of Y this thread is computing
    int k = threadIdx.x + blockIdx.x * blockDim.x;
    int p = threadIdx.y + blockIdx.y * blockDim.y;
    
    if (k < N && p < numPairs) {
        hipFloatComplex sum = make_hipFloatComplex(0, 0);
        
        // Save the value of -2 pi * k / N
        float c = -2 * M_PI * k / N;
        
        // Each thread computes a summation containing N terms
        for (size_t n = 0; n < N; n++) {
            // e^ix = cos x + i sin x
            // Compute x[n] * exp(-2i pi * k * n / N)
            float ti, tr;
            sincosf(c * n, &ti, &tr);
            sum = hipCaddf(sum, hipCmulf(input[p * N + n], make_hipFloatComplex(tr, ti)));
        }
        
        output[p * N + k] = sum;
    }
}

// This function computes the DFT on the GPU.
void performDFT(hipFloatComplex* input, hipFloatComplex* output, uint32_t N, uint32_t numPairs)
{
    hipFloatComplex* d_input;
    hipFloatComplex* d_output;
    
    hipMalloc((void**)&d_output, sizeof(hipFloatComplex) * N * numPairs);
    hipMalloc((void**)&d_input, sizeof(hipFloatComplex) * N * numPairs);
    
    hipMemcpy(d_input, input, sizeof(hipFloatComplex) * N * numPairs, hipMemcpyHostToDevice);
    
    int cuda_device_ix = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, cuda_device_ix);
    
    // One thread for each element of the output vector
    int block_size_x = min(N, prop.maxThreadsDim[0]);
    int block_size_y = min(numPairs, prop.maxThreadsDim[1]);
    int grid_size_x = (N + block_size_x - 1) / block_size_x;
    int grid_size_y = (numPairs + block_size_y - 1) / block_size_y;
    
    dim3 block(block_size_x, block_size_y);
    dim3 grid(grid_size_x, grid_size_y);
    
    dft_kernel<<<grid, block>>>(d_input, d_output, N, numPairs);
    
    hipMemcpy(output, d_output, sizeof(hipFloatComplex) * N * numPairs, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

__global__ void multiplyTwiddleFactors(hipFloatComplex* data, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = tid; i < n / 2; i += stride) {
        float angle = -2 * M_PI * i / n;
        hipFloatComplex twiddle = make_hipFloatComplex(cos(angle), sin(angle));

        hipFloatComplex temp = hipCmulf(data[i + n / 2], twiddle);
        data[i + n / 2] = hipCsubf(data[i], temp);
        data[i] = hipCaddf(data[i], temp);
    }
}

__global__ void bitReversalPermutation(hipFloatComplex* data, int n, int log2n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = tid; i < n; i += stride) {
        int reversedIndex = __brev(i) >> (32 - log2n);
        if (i < reversedIndex) {
            hipFloatComplex temp = data[i];
            data[i] = data[reversedIndex];
            data[reversedIndex] = temp;
        }
    }
}

void multiplyTwiddleFactors(std::vector<hipFloatComplex>& data) {
    int n = data.size();

    for (int i = 0; i < n / 2; i++) {
        int twiddleIndex = i;
        float angle = -2 * M_PI * twiddleIndex / n;
        hipFloatComplex twiddle = make_hipFloatComplex(cos(angle), sin(angle));

        std::cout << "Twiddle factor for index " << i << ": (" << hipCrealf(twiddle) << ", " << hipCimagf(twiddle) << "i)" << std::endl;

        hipFloatComplex temp = hipCmulf(data[i + n / 2], twiddle);
        std::cout << "Multiplied element at index " << i + n / 2 << ": (" << hipCrealf(temp) << ", " << hipCimagf(temp) << "i)" << std::endl;

        data[i + n / 2] = hipCaddf(data[i + n / 2], temp);
        std::cout << "Updated element at index " << i + n / 2 << ": (" << hipCrealf(data[i + n / 2]) << ", " << hipCimagf(data[i + n / 2]) << "i)" << std::endl;

        data[i] = hipCsubf(data[i], temp);
        std::cout << "Updated element at index " << i << ": (" << hipCrealf(data[i]) << ", " << hipCimagf(data[i]) << "i)" << std::endl;

        std::cout << std::endl;
    }
}

std::vector<hipFloatComplex> computeFFT(std::vector<hipFloatComplex>& input) {
    int size = input.size();
    std::vector<hipFloatComplex> output(size);

    hipFloatComplex* d_input;
    hipMalloc(&d_input, size * sizeof(hipFloatComplex));
    hipMemcpy(d_input, input.data(), size * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    std::cout << "Input: ";
    for (int i = 0; i < size; i++) {
        std::cout << "(" << hipCrealf(input[i]) << ", " << hipCimagf(input[i]) << "i) ";
    }
    std::cout << std::endl;

    int log2n = (int)log2(size); //log2 currently must be run on CPU

    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    bitReversalPermutation<<<numBlocks, blockSize>>>(d_input, size, log2n);

    std::vector<hipFloatComplex> bitReversedData(size);
    hipMemcpy(bitReversedData.data(), d_input, size * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    std::cout << "Bit reversal: ";
    for (int i = 0; i < size; i++) {
        std::cout << "(" << hipCrealf(bitReversedData[i]) << ", " << hipCimagf(bitReversedData[i]) << "i) ";
    }
    std::cout << std::endl;

    std::vector<std::vector<hipFloatComplex>> pairs = fftBreakdownHost(bitReversedData);
    std::cout << "FFT Breakdown:" << std::endl;
    for (int i = 0; i < pairs.size(); i++) {
        std::cout << "Pair " << i << ": (" << hipCrealf(pairs[i][0]) << ", " << hipCimagf(pairs[i][0]) << "i) and ("
                  << hipCrealf(pairs[i][1]) << ", " << hipCimagf(pairs[i][1]) << "i)" << std::endl;
    }

    int N = 2;  // Assuming each pair has 2 elements
    int numPairs = pairs.size();

    hipFloatComplex* dftOutput;
    hipMalloc(&dftOutput, N * numPairs * sizeof(hipFloatComplex));

    // Convert the pairs vector to a flattened array
    hipFloatComplex* flatPairs;
    hipMalloc(&flatPairs, N * numPairs * sizeof(hipFloatComplex));
    for (int i = 0; i < numPairs; i++) {
        hipMemcpy(&flatPairs[i * N], pairs[i].data(), N * sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    }

    performDFT(flatPairs, dftOutput, N, numPairs);

    std::cout << "DFT Outputs: ";
    for (int i = 0; i < N * numPairs; i++) {
        hipFloatComplex temp;
        hipMemcpy(&temp, &dftOutput[i], sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
        std::cout << "(" << hipCrealf(temp) << ", " << hipCimagf(temp) << "i) ";
    }
    std::cout << std::endl;

    multiplyTwiddleFactors<<<numBlocks, blockSize>>>(dftOutput, size);

    // Copy the final FFT output back to the host
    hipMemcpy(output.data(), dftOutput, size * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);

    //multiplyTwiddleFactors(output);

    // Free device memory
    hipFree(d_input);
    hipFree(dftOutput);
    hipFree(flatPairs);

    return output;
}

#ifdef COMPILE_MAIN
int main(int argc, char *argv[]) {
    if (argc <= 1 || argc % 2 != 1) {
        std::cout << "Usage: " << argv[0] << " <real1> <imaginary1> <real2> <imaginary2> ... <realN> <imaginaryN>\n";
        return 1;
    }

    int size = (argc - 1) / 2;
    std::vector<hipFloatComplex> complex_values(size);

    for (int i = 0; i < 2 * size; i += 2) {
        int index = i / 2;
        complex_values[index] = make_hipFloatComplex(std::atof(argv[1 + i]), std::atof(argv[2 + i]));
    }

    std::vector<hipFloatComplex> fft_result = computeFFT(complex_values);

    std::cout << "FFT Outputs: ";
    for (int i = 0; i < size; i++) {
        std::cout << "(" << hipCrealf(fft_result[i]) << ", " << hipCimagf(fft_result[i]) << "i) ";
    }
    std::cout << std::endl;

    return 0;
}
#endif
