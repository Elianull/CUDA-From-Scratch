#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

__global__ void linearRegression(int *x_values, int *y_values, float *m_value, float *b_value, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {

    }
}

int main(int argc, char *argv[]) {
    if (argc <= 1 || argc % 2 != 1) {
        printf("Usage: linreg.cu <elem1> <elem2> ... <elemN>\n");
        printf("The number of elements must be even.\n");
        return 1;
    }

    int size = (argc - 1) / 2;
    size_t bytes = size * sizeof(int);

    int *x_values = (int*)malloc(bytes);
    int *y_values = (int*)malloc(bytes);

    for (int i = 0; i < size; i++) {
        x_values[i] = atoi(argv[1 + i]);
        y_values[i] = atoi(argv[1 + size + i]);
    }

    int *d_x_values, *d_y_values;
    float *d_m_value, *d_b_value;

    hipMalloc(&d_x_values, bytes);
    hipMalloc(&d_y_values, bytes);
    hipMalloc(&d_m_value, sizeof(float));
    hipMalloc(&d_b_value, sizeof(float));


    hipMemcpy(d_x_values, x_values, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_y_values, y_values, bytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    linearRegression<<<blocksPerGrid, threadsPerBlock>>>(d_x_values, d_y_values, d_m_value, d_b_value, size);

    float m_value, b_value;
    hipMemcpy(&m_value, d_m_value, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&b_value, d_b_value, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x_values);
    hipFree(d_y_values);
    hipFree(d_m_value);
    hipFree(d_b_value);

    free(x_values);
    free(y_values);

    return 0;
}